
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *a, int *b)
{
    int tx = threadIdx.x;

    switch( tx )
    {
    case 0:
     *a = *a + 10;
     break;
    case 1:
     *b = *b + 3;
     break;
    default:
     *a = *a + 10;
     break;
    }

}

extern "C" void kernel_wrapper(int *a, int *b)
{
    int *d_1, *d_2;

    dim3 threads( 2, 1 );
    dim3 blocks( 1, 1 );    

    hipMalloc( (void **)&d_1, sizeof(int) );
    hipMalloc( (void **)&d_2, sizeof(int) );

    hipMemcpy( d_1, a, sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( d_2, b, sizeof(int), hipMemcpyHostToDevice );

    kernel<<< blocks, threads >>>( d_1, d_2 );

    hipMemcpy( a, d_1, sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( b, d_2, sizeof(int), hipMemcpyDeviceToHost );

    hipFree(d_1);
    hipFree(d_2);
}