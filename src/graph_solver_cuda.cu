
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX 999999

__device__
double minVal(double a, double b) {
    return (a < b) ? a : b;
}

__device__
double fun(int i, int mask, int n, double* dist, double* memo, int* efficientRoute, int* efficientRouteIndex)
{
    if (mask == ((1 << i) | 15)) {        
        efficientRoute[*efficientRouteIndex] = i;
        *efficientRouteIndex = (*efficientRouteIndex + 1) % n;                                                         
        return dist[1 * n + i];    
    }
        
    // memoization
    if (memo[i * (1 << n) + mask] != 0) {
        efficientRoute[*efficientRouteIndex] = i;
        *efficientRouteIndex = (*efficientRouteIndex + 1) % n;                                                         
        return memo[i * (1 << n) + mask];
    }
 
    double res = MAX; // result of this sub-problem
 
    for (int j = 0; j < n; j++)
        if ((mask & (1 << j)) && j != i && j != 1)        
            res = minVal(res, fun(j, mask & (~(1 << i)), n, dist, memo, efficientRoute, efficientRouteIndex)
                                    + dist[i * n + j]);            
            
    return memo[i * (1 << n) + mask] = res;
}

__global__ void kernel(int numStops, int mask, double *dist, double *memo, int *efficientRoute, int *efficientRouteIndex, double *ans)
{    

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;



    for (int i = index; i < numStops; i += stride){
        *ans = minVal(*ans, fun(i, (1 << (numStops + 1)) - 1, numStops, dist, memo, efficientRoute, efficientRouteIndex) + dist[i * numStops + 0]);
    }        
}

extern "C" void kernel_wrapper(int numStops, int mask, double **dist, double **memo, int *efficientRoute)
{
    int *efficientRouteCuda, *efficientRouteIndex;
    double *distCuda, *memoCuda, *ans;

    hipMalloc((void **)&ans, sizeof(double));    
    hipMalloc((void **)&efficientRouteIndex, sizeof(int));
    hipMalloc((void **)&distCuda, numStops * numStops * sizeof(double));
    hipMalloc((void **)&memoCuda, numStops * (1 << numStops) * sizeof(double));
    hipMalloc((void **)&efficientRouteCuda, numStops * sizeof(int));

    int blockSize = 256;
    int numBlocks = 880;

    
    hipMemcpy(distCuda, dist, numStops * numStops * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(memoCuda, memo, numStops * (1 << numStops) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(efficientRouteCuda, efficientRoute, numStops * sizeof(int), hipMemcpyHostToDevice);    

    kernel<<<numBlocks, blockSize>>>(numStops, mask, distCuda, memoCuda, efficientRoute, efficientRouteIndex, ans);

    hipMemcpy(efficientRouteCuda, efficientRoute, numStops * sizeof(int), hipMemcpyDeviceToHost);    

    hipFree(distCuda);
    hipFree(memoCuda);
    hipFree(efficientRouteIndex);
    hipFree(efficientRouteCuda);
}